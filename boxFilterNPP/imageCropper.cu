#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sys/stat.h>
#include <sys/types.h>

// Kernel for converting RGB to Grayscale
__global__ void rgbToGrayKernel(unsigned char* rgb, unsigned char* gray, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        int rgbIdx = idx * 3;
        unsigned char r = rgb[rgbIdx];
        unsigned char g = rgb[rgbIdx + 1];
        unsigned char b = rgb[rgbIdx + 2];
        gray[idx] = 0.299f * r + 0.587f * g + 0.114f * b;
    }
}

// Kernel for cropping the image
__global__ void cropKernel(unsigned char* src, unsigned char* dst, int srcWidth, int srcHeight, int dstWidth, int dstHeight, int offsetX, int offsetY) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < dstWidth && y < dstHeight) {
        int srcIdx = (y + offsetY) * srcWidth + (x + offsetX);
        int dstIdx = y * dstWidth + x;
        dst[dstIdx] = src[srcIdx];
    }
}

// Kernel for adjusting shadows and highlights
__global__ void adjustShadowsHighlightsKernel(unsigned char* img, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        unsigned char pixel = img[idx];
        if (pixel <= 128) {
            img[idx] = static_cast<unsigned char>(min(255.0f, pixel * 1.1f)); // Increase shadows by 10%
        } else {
            img[idx] = static_cast<unsigned char>(max(0.0f, pixel * 1.05f)); // increase highlights by 5%
        }
    }
}

// Function to load a PPM image (color)
void loadPPM(const std::string& filename, unsigned char*& data, int& width, int& height) {
    std::ifstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        throw std::runtime_error("Unable to open file " + filename);
    }

    std::string magic;
    file >> magic;
    if (magic != "P6") {
        throw std::runtime_error("Invalid PPM file");
    }

    file >> width >> height;

    int maxVal;
    file >> maxVal;
    file.ignore(256, '\n'); // Skip to the next line

    data = new unsigned char[width * height * 3];
    file.read(reinterpret_cast<char*>(data), width * height * 3);

    file.close();
}

// Function to save a PGM image
void savePGM(const std::string& filename, unsigned char* data, int width, int height) {
    std::ofstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        throw std::runtime_error("Unable to open file " + filename);
    }

    file << "P5\n" << width << " " << height << "\n255\n";
    file.write(reinterpret_cast<char*>(data), width * height);

    file.close();
}

int main(int argc, char* argv[]) {
    printf("%s Starting...\n\n", argv[0]);

    try {
        std::vector<std::string> inputFiles = {
            "img1.ppm", "img2.ppm", "img3.ppm", "img4.ppm", "img5.ppm",
            "img6.ppm", "img7.ppm", "img8.ppm", "img9.ppm", "img10.ppm"
        };

        std::string outputDir = "processed_images";
        // Create the output directory
        #if defined(_WIN32)
            _mkdir(outputDir.c_str());
        #else 
            mkdir(outputDir.c_str(), 0755);
        #endif

        for (const auto& sFilename : inputFiles) {
            unsigned char* h_rgb = nullptr;
            int width, height;
            loadPPM(sFilename, h_rgb, width, height);

            unsigned char* h_gray = new unsigned char[width * height];
            unsigned char* d_rgb;
            unsigned char* d_gray;

            checkCudaErrors(hipMalloc(&d_rgb, width * height * 3 * sizeof(unsigned char)));
            checkCudaErrors(hipMalloc(&d_gray, width * height * sizeof(unsigned char)));

            checkCudaErrors(hipMemcpy(d_rgb, h_rgb, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

            dim3 blockSize(16, 16);
            dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
            rgbToGrayKernel<<<gridSize, blockSize>>>(d_rgb, d_gray, width, height);

            checkCudaErrors(hipMemcpy(h_gray, d_gray, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));

            int shorterSide = std::min(width, height);
            int longerSide = std::max(width, height);
            int offset = (longerSide - shorterSide) / 2;

            int dstWidth = shorterSide;
            int dstHeight = shorterSide;

            unsigned char* h_dst = new unsigned char[dstWidth * dstHeight];
            unsigned char* d_dst;

            checkCudaErrors(hipMalloc(&d_dst, dstWidth * dstHeight * sizeof(unsigned char)));
            checkCudaErrors(hipMemcpy(d_gray, h_gray, width * height * sizeof(unsigned char), hipMemcpyHostToDevice));

            cropKernel<<<gridSize, blockSize>>>(d_gray, d_dst, width, height, dstWidth, dstHeight, offset, 0);

            // Adjust shadows and highlights
            adjustShadowsHighlightsKernel<<<gridSize, blockSize>>>(d_dst, dstWidth, dstHeight);

            checkCudaErrors(hipMemcpy(h_dst, d_dst, dstWidth * dstHeight * sizeof(unsigned char), hipMemcpyDeviceToHost));

            std::string outputFilename = outputDir + "/" + sFilename.substr(0, sFilename.find_last_of('.')) + "_cropped_adjusted.pgm";
            savePGM(outputFilename, h_dst, dstWidth, dstHeight);
            std::cout << "Saved image: " << outputFilename << std::endl;

            delete[] h_rgb;
            delete[] h_gray;
            delete[] h_dst;
            hipFree(d_rgb);
            hipFree(d_gray);
            hipFree(d_dst);
        }

        exit(EXIT_SUCCESS);
    } catch (std::exception& e) {
        std::cerr << "Program error! The following exception occurred: \n" << e.what() << std::endl;
        exit(EXIT_FAILURE);
    } catch (...) {
        std::cerr << "Program error! An unknown type of exception occurred. \n";
        exit(EXIT_FAILURE);
    }

    return 0;
}
